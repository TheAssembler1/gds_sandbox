#include <stdio.h> 
#include <stdlib.h>
#include <string.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <unistd.h>
#include <cufile.h>
#include <hip/hip_runtime.h>

/*
 * Best Practice: https://docs.nvidia.com/gpudirect-storage/best-practices-guide
 * API Reference: https://docs.nvidia.com/gpudirect-storage/api-reference-guide/index.html
 * API Reference PDF: https://docs.nvidia.com/cuda/archive/11.6.0/pdf/cuFile_API.pdf
 */

#define USAGE_DETAILS \
  "Arguments:\n" \
  "  <gen_files>  : (true or false)  Set to true to generate files, false otherwise\n" \
  "  <file>       : (small_files, big_files)  Specify file type\n" \
  "  <dir>        : (single_dir, many_dir)  Specify directory type"

#define DATA_OUTPUT_DIR "./data"
#define FILE_PREFIX "/file_"
#define FILE_SUFFIX ".data"

#undef DEBUG_KERNEL_FUNC

// FIXME: this should be a cmd arg
unsigned int num_files = 100;
unsigned int small_file_size_bytes = 512;
unsigned int big_file_size_bytes = 4096;

void validate_args(const char* gen_files, const char* file);
void create_data(const char* file);
void run_gpu_operations();

// FIXME: this should be a cmd arg
#define DATA_PER_THREAD 128
#define THREADS_PER_BLOCK 256

__global__ void simple_gpu_kernel(char* data, size_t size) {
  size_t index = blockIdx.x * blockDim.x + threadIdx.x;
#ifdef DEBUG_KERNEL_FUNC
  printf("starting simple_gpu_kernel with index %d\n", index);
#endif

  if (index * DATA_PER_THREAD < size) {
    size_t end_index = min((index + 1) * DATA_PER_THREAD, size);
    for (size_t i = index * DATA_PER_THREAD; i < end_index; i++) {
#ifdef DEBUG_KERNEL_FUNC
        printf("%c", data[i]);
#endif
    }
  }
}

int main(int argc, char** argv) {
  if(argc < 3) {
    printf("%s\n", USAGE_DETAILS);
    exit(1);
  }
  
  /* grab cmd args */
  const char* gen_files = argv[1];
  const char* file = argv[2];

  validate_args(gen_files, file);

  /* check and generate files and folders */
  if(!strcmp(gen_files, "true")) {
    printf("generating test data\n");
    create_data(file);
  } else {
    printf("assuming test data is already generated\n");
  }

  run_gpu_operations();

  return 0;
}

// FIXME: this should be a parameter
#define DATA_MOVEMENT_TYPE "posix"
#define DATA_MOVEMENT_OP "read"

typedef enum {
  READ,
  INVALID_DATA_MOVEMENT_OP
} data_movement_op_t;

typedef enum {
  MALLOC,
  INVALID_DATA_MOVEMENT_TYPE
} data_movement_type_t;

// FIXME: file size
size_t file_size = 0;
void *device_data = NULL;

static void gpu_read_malloc_data(char* file_path) {
  int fd;
  void *host_data = NULL;

  fd = open(file_path, O_RDONLY);
  if (fd == -1) {
      perror("error opening file");
      exit(1);
  }

  file_size = lseek(fd, 0, SEEK_END);
  if (file_size == -1) {
      perror("error getting file size");
      close(fd);
      exit(1);
  }
  lseek(fd, 0, SEEK_SET);

  host_data = malloc(file_size);
  if (host_data == NULL) {
      perror("error allocating memory for file data");
      close(fd);
      exit(1);
  }

  ssize_t bytes_read = read(fd, host_data, file_size);
  if (bytes_read != file_size) {
      perror("Error reading file");
      free(host_data);
      close(fd);
      exit(1);
  }

  hipError_t cuda_status = hipMalloc(&device_data, file_size);
  if (cuda_status != hipSuccess) {
      printf("failed CUDA malloc: %s\n", hipGetErrorString(cuda_status));
      free(host_data);
      close(fd);
      exit(1);
  }

  cuda_status = hipMemcpy(device_data, host_data, file_size, hipMemcpyHostToDevice);
  if (cuda_status != hipSuccess) {
      printf("CUDA memcpy failed: %s\n", hipGetErrorString(cuda_status));
      hipFree(device_data);
      free(host_data);
      close(fd);
      exit(1);
  }

  free(host_data);
  close(fd);
}

void run_gpu_operations() {
  CUfileError_t file_status;
  hipError_t cuda_status;
  data_movement_op_t data_movement_op = INVALID_DATA_MOVEMENT_OP;
  data_movement_type_t data_movement_type = INVALID_DATA_MOVEMENT_TYPE;

  /* initialize  state of critical performance path */
  file_status = cuFileDriverOpen();
  if(file_status.err != CU_FILE_SUCCESS) {
    printf("failed to initialize cuFileDriver\n");
    exit(1);
  } else {
    printf("successfully initialize cuFileDriver\n");
  }

  printf("data movement type: %s, operation: %s\n", DATA_MOVEMENT_TYPE, DATA_MOVEMENT_OP);

  if(!strcmp(DATA_MOVEMENT_TYPE, "posix")) {
    data_movement_type = MALLOC;
  } else {
    printf("invalid data movement type\n");
    exit(1);
  }

  if(!strcmp(DATA_MOVEMENT_OP, "read")) {
    data_movement_op = READ;
  } else {
    printf("invalid data movement operation\n");
    exit(1);
  }

  for(int i = 1; i <= num_files; i++) {
    /* get file name */
    char file_num_str[32];
    sprintf(file_num_str, "%d", i);
    size_t file_name_len = strlen(DATA_OUTPUT_DIR) + strlen(FILE_PREFIX) + strlen(file_num_str) + strlen(FILE_SUFFIX)  + 1;
    char* file_name = (char*)malloc(file_name_len * sizeof(char));
  
    strcpy(file_name, DATA_OUTPUT_DIR);
    strcat(file_name, FILE_PREFIX);
    strcat(file_name, file_num_str);
    strcat(file_name, FILE_SUFFIX);
  
    printf("reading file %s\n", file_name);

    /* select data movement operation */
    switch(data_movement_op) {
      case MALLOC:
        switch(data_movement_type) {
          case READ:
            gpu_read_malloc_data(file_name);
            break;
          case INVALID_DATA_MOVEMENT_TYPE:
            exit(1);
            break;
          default:
            exit(1);
            break;
        }
      break;
      case INVALID_DATA_MOVEMENT_OP:
        exit(1);
        break;
      default:
        exit(1);
        break;
    }

    /* run gpu kernel */
    size_t block_size = THREADS_PER_BLOCK;  
    size_t grid_size = (file_size + DATA_PER_THREAD * block_size - 1) / (DATA_PER_THREAD * block_size);
    printf("block size: %d, grid_size: %d\n", block_size, grid_size);
    simple_gpu_kernel<<<grid_size, block_size>>>((char*)device_data, file_size);

    cuda_status = hipGetLastError();
    if (cuda_status != hipSuccess) {
        printf("failed CUDA kernel launch: %s\n", hipGetErrorString(cuda_status));
    } else {
        printf("successfully launched kernel.\n");
    }

    cuda_status = hipDeviceSynchronize();
    if(cuda_status != hipSuccess) {
      const char* error_string = hipGetErrorString(cuda_status);
      printf("error found when syncing device: %s\n", error_string);
    } else {
      printf("no errors when syncing device\n");
    }

    /* free memory buffer on GPU device */
    hipFree(device_data);
  }

  cuFileDriverClose();
}

static void create_directory(const char* dir) {
  struct stat dir_stat;
  if(stat(dir, &dir_stat) == -1) {
    printf("creating directory %s\n", dir);

    if(mkdir(dir, 0777) != 0) {
      printf("failed to create directory %s", dir);
      exit(1);
    }
  } else {
    printf("directory %s already exists... skipping creation\n", dir);
  }
}

static void create_file(const char* file, int file_num) {
  char file_num_str[32];
  sprintf(file_num_str, "%d", file_num);
  size_t file_name_len = strlen(DATA_OUTPUT_DIR) + strlen(FILE_PREFIX) + strlen(file_num_str) + strlen(FILE_SUFFIX)  + 1;
  char* file_name = (char*)malloc(file_name_len * sizeof(char));

  strcpy(file_name, DATA_OUTPUT_DIR);
  strcat(file_name, FILE_PREFIX);
  strcat(file_name, file_num_str);
  strcat(file_name, FILE_SUFFIX);

  printf("creating file %s\n", file_name);

  FILE* fp = fopen(file_name, "wb");
  if(fp == NULL) {
    printf("failed to open file %s\n", file_name);
    exit(1);
  }
  
  int data_size_bytes = small_file_size_bytes;
  if(!strcmp(file, "big_files")) {
    data_size_bytes = big_file_size_bytes;
  }

  char* data = (char*)malloc(data_size_bytes * sizeof(char));
  size_t wb = fwrite(data, sizeof(char), data_size_bytes, fp);
  if(wb != data_size_bytes) {
    printf("failed to write data file %s\n", file_name);
    exit(1);
  }

  fclose(fp);

  free(data);
  free(file_name);
}

void create_data(const char* file) {
  /* create main data directory */
  create_directory(DATA_OUTPUT_DIR);
  
  for(int i = 1; i <= num_files; i++) {
    create_file(file, i);
  }
}

void validate_args(const char* gen_files, const char* file) {
  if(strcmp(gen_files, "true") && strcmp(gen_files, "false")) {
    printf("invalid gen_files\n");
    printf("%s\n", USAGE_DETAILS);
    exit(1);
  } 

  if(strcmp(file, "small_files") && strcmp(file, "big_files")) {
    printf("invalid file\n");
    printf("%s\n", USAGE_DETAILS);
    exit(1);
  }
}
