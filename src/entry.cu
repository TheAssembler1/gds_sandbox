#include <stdio.h> 
#include <stdlib.h>
#include <string.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <unistd.h>
#include <sys/time.h>
#include <sys/mman.h>
#include <cufile.h>
#include <hip/hip_runtime.h>

/*
 * Best Practice: https://docs.nvidia.com/gpudirect-storage/best-practices-guide
 * API Reference: https://docs.nvidia.com/gpudirect-storage/api-reference-guide/index.html
 * API Reference PDF: https://docs.nvidia.com/cuda/archive/11.6.0/pdf/cuFile_API.pdf
 */

#define USAGE_DETAILS \
  "Arguments:\n" \
  "  <gen_files>  : (true or false)  Set to true to generate files, false otherwise\n" \
  "  <file>       : (small_files, big_files)  Specify file type\n" \
  "  <dir>        : (single_dir, many_dir)  Specify directory type" \
  "  <data_movement_type>        : (posix, gpu_direct, mmap) How to move data between GPU, CPU, and storage" \
  "  <num_files>        : (single_dir, many_dir)  Specify number of files to operate on"

#define DATA_OUTPUT_DIR "./data"
#define FILE_PREFIX "/file_"
#define FILE_SUFFIX ".data"

#define STATUS_UPDATE_FILE_NUM 100

#define PROFILE_PREFIX "PROFILE INFO: "

// NOTE: printing macros
#undef DEBUG_KERNEL_FUNC
#define CPU_DEBUG
#undef CPU_STATUS_DEBUG

#ifdef CPU_DEBUG
    #define cpu_printf(f_, ...) printf((f_), ##__VA_ARGS__)
#else
    #define cpu_printf(f_, ...) (void)0
#endif

#ifdef CPU_STATUS_DEBUG
    #define cpu_status_printf(f_, ...) printf((f_), ##__VA_ARGS__)
#else
    #define cpu_status_printf(f_, ...) (void)0
#endif

#ifdef GPU_DEBUG
    #define gpu_printf(f_, ...) printf((f_), ##__VA_ARGS__)
#else
    #define gpu_printf(f_, ...) /* Do nothing */
#endif


// FIXME: this should be a cmd arg
unsigned long num_files = 10;
unsigned long big_file_size_bytes = 136870912;
unsigned long small_file_size_bytes = 512;

void validate_args(const char* gen_files, const char* file);
void create_data(const char* file);
void run_gpu_operations();

// FIXME: this should be a cmd arg
#define DATA_PER_THREAD 128
#define THREADS_PER_BLOCK 256

char* __data_movement_type;

// FIXME: this should be a parameter
#define DATA_MOVEMENT_TYPE __data_movement_type
#define DATA_MOVEMENT_OP "read"

#define TIME_FUNC_RET(func, inc_var, ret) { \
  struct timeval start, end; \
  gettimeofday(&start, NULL); \
  ret = func; \
  gettimeofday(&end, NULL); \
  long double elapsed_time = (end.tv_sec - start.tv_sec) * 1000.0 + \
                             (end.tv_usec - start.tv_usec) / 1000.0; \
  inc_var += elapsed_time; \
}

#define TIME_FUNC(func, inc_var) { \
  struct timeval start, end; \
  gettimeofday(&start, NULL); \
  func; \
  gettimeofday(&end, NULL); \
  long double elapsed_time = (end.tv_sec - start.tv_sec) * 1000.0 + \
                             (end.tv_usec - start.tv_usec) / 1000.0; \
  inc_var += elapsed_time; \
}

long double total_data_movement_time = 0;
#define TIME_DATA_MOVEMENT_FUNC(func) TIME_FUNC(func, total_data_movement_time)
#define TIME_DATA_MOVEMENT_FUNC_RET(func, ret) TIME_FUNC_RET(func, total_data_movement_time, ret)

long double total_gpu_func_time = 0;
#define TIME_GPU_EXECUTION_FUNC(func) TIME_FUNC(func, total_gpu_func_time)
#define TIME_GPU_EXECUTION_FUNC_RET(func, ret) TIME_FUNC_RET(func, total_gpu_func_time, ret)

long double total_metadata_time = 0;
#define TIME_METADATA_FUNC(func) TIME_FUNC(func, total_metadata_time)
#define TIME_METADATA_FUNC_RET(func, ret) TIME_FUNC_RET(func, total_metadata_time, ret)

__global__ void simple_gpu_kernel(char* data, size_t size) {
  size_t index = blockIdx.x * blockDim.x + threadIdx.x;
  gpu_printf("starting simple_gpu_kernel with index %d\n", index);

  if (index * DATA_PER_THREAD < size) {
    size_t end_index = min((index + 1) * DATA_PER_THREAD, size);
    for (size_t i = index * DATA_PER_THREAD; i < end_index; i++) {
      gpu_printf("%c", data[i]);
    }
  }
}

static int run(int argc, char** argv) {
  if(argc < 5) {
    printf("%s\n", USAGE_DETAILS);
    exit(1);
  }
  
  /* grab cmd args */
  const char* gen_files = argv[1];
  const char* file = argv[2];

  // FIXME: hackey way to get args and not validating them
  __data_movement_type = argv[3];
  num_files = atoi(argv[4]);

  validate_args(gen_files, file);

  /* check and generate files and folders */
  if(!strcmp(gen_files, "true")) {
    printf("generating test data with %lu files each files is %lu bytes\n", num_files, big_file_size_bytes);
    create_data(file);
    return 0;
  } else {
    printf("assuming test data is already generated\n");
  }

  /* config info should not be hidden behind debug */
  // FIXME: assumes big files
  printf("total files: %lu\n", num_files);
  printf("size of each file: %lu bytes\n", big_file_size_bytes);
  printf("total data movement size: %lu bytes %.2f megabytes\n", 
    num_files * big_file_size_bytes, 
    (num_files * big_file_size_bytes) / (1024.0 * 1024));
  printf("data movement operation: %s, data movement type %s\n", DATA_MOVEMENT_OP, DATA_MOVEMENT_TYPE);

  run_gpu_operations();

  return 0;
}

int main(int argc, char** argv) {
  long double total_runtime = 0;
  int ret;
  TIME_FUNC_RET(run(argc, argv), total_runtime, ret);

  /* profiling info should not be hidden behind debug */
  printf("%s total time: %Lf ms\n", PROFILE_PREFIX, total_runtime);
  printf("%s total data movement time: %Lf ms\n", PROFILE_PREFIX, total_data_movement_time);
  printf("%s total gpu function execution time: %Lf ms\n", PROFILE_PREFIX, total_gpu_func_time);
  printf("%s total metadata time: %Lf ms\n", PROFILE_PREFIX, total_metadata_time);
  printf("%s average data movement per file time %Lf ms\n", PROFILE_PREFIX, total_data_movement_time / num_files);
  printf("%s average gpu function execution time: %Lf ms\n", PROFILE_PREFIX, total_gpu_func_time / num_files);

  return ret;
}

typedef enum {
  READ,
  INVALID_DATA_MOVEMENT_OP
} data_movement_op_t;

typedef enum {
  MALLOC,
  GPU_DIRECT,
  MMAP,
  INVALID_DATA_MOVEMENT_TYPE
} data_movement_type_t;

// FIXME: file size
size_t file_size = 0;
void *device_data = NULL;

static void gpu_read_direct_data(char* filepath, int file_num) {
  int fd;
  ssize_t ret;
  size_t buff_size;
  CUfileError_t status;
  CUfileDescr_t cf_descr;
  CUfileHandle_t cf_handle;

  TIME_METADATA_FUNC_RET(open(filepath, O_RDONLY | O_DIRECT), fd);
  if (fd < 0) {
      perror("File open failed");
      return;
  }

  TIME_METADATA_FUNC_RET(lseek(fd, 0, SEEK_END), file_size);
  if (file_size == -1) {
      perror("File size retrieval failed");
      close(fd);
      return;
  }
  TIME_METADATA_FUNC(lseek(fd, 0, SEEK_SET));

  buff_size = (size_t)file_size;
  memset(&cf_descr, 0, sizeof(CUfileDescr_t));
  cf_descr.handle.fd = fd;
  cf_descr.type = CU_FILE_HANDLE_TYPE_OPAQUE_FD;
  status = cuFileHandleRegister(&cf_handle, &cf_descr);
  if (status.err != CU_FILE_SUCCESS) {
      fprintf(stderr, "cuFileHandleRegister failed\n");
      cuFileDriverClose();
      close(fd);
      exit(1);
  }

  hipError_t cuda_result = hipMalloc(&device_data, buff_size);
  if (cuda_result != hipSuccess) {
      fprintf(stderr, "CUDA malloc failed: %s\n", hipGetErrorString(cuda_result));
      cuFileHandleDeregister(cf_handle);
      cuFileDriverClose();
      close(fd);
      exit(1);
  }

  status = cuFileBufRegister(device_data, buff_size, 0);
  if (status.err != CU_FILE_SUCCESS) {
      fprintf(stderr, "buffer registration failed with error code %d\n", status.err);
    
      hipFree(device_data);
      cuFileHandleDeregister(cf_handle);
      cuFileDriverClose();
      close(fd);
      exit(1);
  }

  ret = cuFileRead(cf_handle, device_data, buff_size, 0, 0);
  if (ret < 0) {
      fprintf(stderr, "cuFileRead failed: %ld\n", ret);
      exit(1);
  }

  status = cuFileBufDeregister(device_data);
  if (status.err != CU_FILE_SUCCESS) {
      fprintf(stderr, "Buffer deregistration failed\n");
      exit(1);
  }

  cuFileHandleDeregister(cf_handle);
  close(fd);
}

static void gpu_read_mmap_data(char* file_path, int file_num) {
  int fd;
  void *host_data = NULL;

  TIME_METADATA_FUNC_RET(open(file_path, O_RDONLY), fd);
  if (fd == -1) {
      perror("error opening file");
      exit(1);
  }

  TIME_METADATA_FUNC_RET(lseek(fd, 0, SEEK_END), file_size);
  if (file_size == -1) {
      perror("error getting file size");
      close(fd);
      exit(1);
  }
  TIME_METADATA_FUNC(lseek(fd, 0, SEEK_SET));

  host_data = mmap(NULL, file_size, PROT_READ, MAP_PRIVATE, fd, 0);
  if (host_data == MAP_FAILED) {
      perror("error mapping file into memory");
      close(fd);
      exit(1);
  }

  hipError_t cuda_status = hipMalloc(&device_data, file_size);
  if (cuda_status != hipSuccess) {
      cpu_printf("failed CUDA malloc: %s\n", hipGetErrorString(cuda_status));
      munmap(host_data, file_size); 
      close(fd);
      exit(1);
  }

  cuda_status = hipMemcpy(device_data, host_data, file_size, hipMemcpyHostToDevice);
  if (cuda_status != hipSuccess) {
      cpu_printf("CUDA memcpy failed: %s\n", hipGetErrorString(cuda_status));
      hipFree(device_data);
      munmap(host_data, file_size);
      close(fd);
      exit(1);
  }

  munmap(host_data, file_size);
  close(fd);
}

static void gpu_read_malloc_data(char* file_path, int file_num) {
  int fd;
  void *host_data = NULL;

  TIME_METADATA_FUNC_RET(open(file_path, O_RDONLY), fd);
  if (fd == -1) {
      perror("error opening file");
      exit(1);
  }

  TIME_METADATA_FUNC_RET(lseek(fd, 0, SEEK_END), file_size);
  if (file_size == -1) {
      perror("error getting file size");
      close(fd);
      exit(1);
  }
  TIME_METADATA_FUNC(lseek(fd, 0, SEEK_SET));

  host_data = malloc(file_size);
  if (host_data == NULL) {
      perror("error allocating memory for file data");
      close(fd);
      exit(1);
  }

  unsigned long bytes_read = read(fd, host_data, file_size);
  printf("bytes read: %lu, file size: %lu\n", bytes_read, file_size);
  if (bytes_read != file_size) {
      perror("error reading file");
      free(host_data);
      close(fd);
      exit(1);
  }

  hipError_t cuda_status = hipMalloc(&device_data, file_size);
  if (cuda_status != hipSuccess) {
    cpu_printf("failed CUDA malloc: %s\n", hipGetErrorString(cuda_status));
      free(host_data);
      close(fd);
      exit(1);
  }

  cuda_status = hipMemcpy(device_data, host_data, file_size, hipMemcpyHostToDevice);
  if (cuda_status != hipSuccess) {
      cpu_printf("CUDA memcpy failed: %s\n", hipGetErrorString(cuda_status));
      hipFree(device_data);
      free(host_data);
      close(fd);
      exit(1);
  }

  free(host_data);
  close(fd);
}

static void exec_gpu_function() {
    hipError_t cuda_status;

    /* run gpu kernel */
    size_t block_size = THREADS_PER_BLOCK;  
    size_t grid_size = (file_size + DATA_PER_THREAD * block_size - 1) / (DATA_PER_THREAD * block_size);
    cpu_printf("block size: %d, grid_size: %d\n", block_size, grid_size);
    simple_gpu_kernel<<<grid_size, block_size>>>((char*)device_data, file_size);

    cuda_status = hipGetLastError();
    if (cuda_status != hipSuccess) {
      cpu_printf("failed CUDA kernel launch: %s\n", hipGetErrorString(cuda_status));
    } else {
      cpu_printf("successfully launched kernel.\n");
    }

    cuda_status = hipDeviceSynchronize();
    if(cuda_status != hipSuccess) {
      const char* error_string = hipGetErrorString(cuda_status);
      cpu_printf("error found when syncing device: %s\n", error_string);
    } else {
      cpu_printf("no errors when syncing device\n");
    }
}

void run_gpu_operations() {
  CUfileError_t file_status;
  data_movement_op_t data_movement_op = INVALID_DATA_MOVEMENT_OP;
  data_movement_type_t data_movement_type = INVALID_DATA_MOVEMENT_TYPE;

  /* initialize  state of critical performance path */
  file_status = cuFileDriverOpen();
  if(file_status.err != CU_FILE_SUCCESS) {
    cpu_printf("failed to initialize cuFileDriver\n");
    exit(1);
  } else {
    cpu_printf("successfully initialize cuFileDriver\n");
  }

  cpu_printf("data movement type: %s, operation: %s\n", DATA_MOVEMENT_TYPE, DATA_MOVEMENT_OP);

  if(!strcmp(DATA_MOVEMENT_TYPE, "posix")) {
    data_movement_type = MALLOC;
  } else if(!strcmp(DATA_MOVEMENT_TYPE, "gpu_direct")) {
    data_movement_type = GPU_DIRECT;
  } else if(!strcmp(DATA_MOVEMENT_TYPE, "mmap")) {
    data_movement_type = MMAP;
  } else {
    cpu_printf("invalid data movement type\n");
    exit(1);
  }

  if(!strcmp(DATA_MOVEMENT_OP, "read")) {
    data_movement_op = READ;
  } else {
    cpu_printf("invalid data movement operation\n");
    exit(1);
  }

  for(int i = 1; i <= num_files; i++) {
    if(i % STATUS_UPDATE_FILE_NUM == 0) {
      cpu_status_printf("processed %d files\n", i);
    }

    /* get file name */
    char file_num_str[32];
    sprintf(file_num_str, "%d", i);
    size_t file_name_len = strlen(DATA_OUTPUT_DIR) + strlen(FILE_PREFIX) + strlen(file_num_str) + strlen(FILE_SUFFIX)  + 1;
    char* file_name = (char*)malloc(file_name_len * sizeof(char));
  
    strcpy(file_name, DATA_OUTPUT_DIR);
    strcat(file_name, FILE_PREFIX);
    strcat(file_name, file_num_str);
    strcat(file_name, FILE_SUFFIX);
  
    cpu_printf("reading file %s\n", file_name);

    /* select data movement operation */
    switch(data_movement_op) {
      case READ:
        switch(data_movement_type) {
          case MALLOC:
            TIME_DATA_MOVEMENT_FUNC(gpu_read_malloc_data(file_name, i));
            break;
          case GPU_DIRECT:
            TIME_DATA_MOVEMENT_FUNC(gpu_read_direct_data(file_name, i));
            break;
          case MMAP:
            TIME_DATA_MOVEMENT_FUNC(gpu_read_mmap_data(file_name, i));
            break;
          case INVALID_DATA_MOVEMENT_TYPE:
            exit(1);
            break;
          default:
            exit(1);
            break;
        }
      break;
      case INVALID_DATA_MOVEMENT_OP:
        exit(1);
        break;
      default:
        exit(1);
        break;
    }

    /* execute the gpu function */
    TIME_GPU_EXECUTION_FUNC(exec_gpu_function());

    /* free memory buffer on GPU device */
    hipFree(device_data);
  }

  cuFileDriverClose();
}

static void create_directory(const char* dir) {
  struct stat dir_stat;
  if(stat(dir, &dir_stat) == -1) {
    cpu_printf("creating directory %s\n", dir);

    if(mkdir(dir, 0777) != 0) {
      cpu_printf("failed to create directory %s", dir);
      exit(1);
    }
  } else {
    cpu_printf("directory %s already exists... skipping creation\n", dir);
  }
}

static void create_file(const char* file, int file_num) {
  char file_num_str[32];
  sprintf(file_num_str, "%d", file_num);
  size_t file_name_len = strlen(DATA_OUTPUT_DIR) + strlen(FILE_PREFIX) + strlen(file_num_str) + strlen(FILE_SUFFIX)  + 1;
  char* file_name = (char*)malloc(file_name_len * sizeof(char));

  strcpy(file_name, DATA_OUTPUT_DIR);
  strcat(file_name, FILE_PREFIX);
  strcat(file_name, file_num_str);
  strcat(file_name, FILE_SUFFIX);

  cpu_printf("creating file %s\n", file_name);

  FILE* fp = fopen(file_name, "wb");
  if(fp == NULL) {
    cpu_printf("failed to open file %s\n", file_name);
    exit(1);
  }
  
  unsigned long data_size_bytes = small_file_size_bytes;
  if(!strcmp(file, "big_files")) {
    data_size_bytes = big_file_size_bytes;
  }

  char* data = (char*)malloc(data_size_bytes * sizeof(char));
  size_t wb = fwrite(data, sizeof(char), data_size_bytes, fp);
  if(wb != data_size_bytes) {
    perror("fwrite");
    exit(1);
  }

  fclose(fp);

  free(data);
  free(file_name);
}

void create_data(const char* file) {
  /* create main data directory */
  create_directory(DATA_OUTPUT_DIR);
  
  for(int i = 1; i <= num_files; i++) {
    if(i % STATUS_UPDATE_FILE_NUM == 0) {
      cpu_status_printf("created %d files\n", i);
    }
    create_file(file, i);
  }
}

void validate_args(const char* gen_files, const char* file) {
  if(strcmp(gen_files, "true") && strcmp(gen_files, "false")) {
    cpu_printf("invalid gen_files\n");
    cpu_printf("%s\n", USAGE_DETAILS);
    exit(1);
  } 

  if(strcmp(file, "small_files") && strcmp(file, "big_files")) {
    cpu_printf("invalid file\n");
    cpu_printf("%s\n", USAGE_DETAILS);
    exit(1);
  }
}
